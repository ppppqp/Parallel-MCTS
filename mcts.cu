#include "hip/hip_runtime.h"
#include "mcts.h"
#include "game.h"
#include <stack>
#include <algorithm>    
#include <random>       
#include <hiprand/hiprand_kernel.h>
using namespace std;

#define MAX_SIM_STEP 100
#define MAX_EXPAND_STEP 100

#define BOARD_W 9
#define BLOCK_W 9

#define D_NONE 0
#define D_WHITE 1
#define D_BLACK 2

// act:
//  15:8 = x
//   7:0 = y
// 0xFFFF: no act is available
__device__ uint16_t git_random_action(uint16_t *actions, int *actions_len){
    if (*actions_len == 0) return 0xFFFFU;
    if (*actions_len == 1) {
        *actions_len--;
        return actions[0];
    }
    uint16_t act = 0xFFFFU;

    hiprandState state;
    hiprand_init((unsigned int)clock64(), threadIdx.y * blockDim.x + threadIdx.x, 0, &state);

    int rand_idx = (int)(hiprand_uniform(&state) * actions_len);
    rand_idx = rand_idx >= actions_len ? actions_len : rand_idx;

    act = actions[rand_idx];
    for (int i = rand_idx; i < actions_len-1; ++i) {
        actions[i] = actions[i+1];
    }

    *actions_len--;
    return act;
}



__device__ void update_board(uint8_t *s_board, uint8_t act_x, uint8_t act_y, ROLE *role){
    uint8_t myStone = (*role == ROLE::BLACK) ? D_BLACK : D_WHITE;
    uint8_t opponentStone = (*role == ROLE::BLACK) ? D_WHITE : D_BLACK;

    uint8_t y = 0;
    uint8_t x = 0;

    // top
    y = act_y-1;
    x = act_x;
    while(y >= 0 && s_board[y * BOARD_W + x] == opponentStone){
        y--;
    }
    if(y >= 0 && s_board[y * BOARD_W + x] == myStone){
        for(int i = act_y-1; i > y; i--){
            s_board[i * BOARD_W + x] = myStone;
        }
    }

    // bottom
    y = act_y+1;
    x = act_x;
    while(y < BOARD_W && s_board[y * BOARD_W + x] == opponentStone){
        y++;
    }
    if(y < BOARD_W && s_board[y * BOARD_W + x] == myStone){
        for(int i = act_y+1; i < y; i++){
            s_board[i * BOARD_W + x] = myStone;
        }
    }

    // right
    y = act_y;
    x = act_x+1;
    while(x < BOARD_W && s_board[y * BOARD_W + x] == opponentStone){
        x++;
    }
    if(x < BOARD_W && s_board[y * BOARD_W + x] == myStone){
        for(int i = act_x+1; i < x; i++){
            s_board[y * BOARD_W + i] = myStone;
        }
    }

    // left
    y = act_y;
    x = act_x-1;
    while(x >= 0 && s_board[y * BOARD_W + x] == opponentStone){
        x--;
    }
    if(x >= 0 && s_board[y * BOARD_W + x] == myStone){
        for(int i = act_x-1; i > x; i--){
            s_board[y * BOARD_W + i] = myStone;
        }
    }

    // top-left
    y = act_y-1;
    x = act_x-1;
    int count = 0;
    while(x >= 0 && y >= 0 && s_board[y * BOARD_W + x] == opponentStone){
        x--;
        y--;
        count ++;
    }
    if(x >=0 && y >= 0 && s_board[y * BOARD_W + x] == myStone){
        for(int i = 0; i < count; i++){
            s_board[(act_y-1-i) * BOARD_W + act_x-1-i] = myStone;
        }
    }

    // top-right
    y = act_y-1;
    x = act_x+1;
    count = 0;
    while(x < BOARD_W && y >= 0 && s_board[y * BOARD_W + x] == opponentStone){
        x++;
        y--;
        count ++;
    }
    if(x < BOARD_W && y >= 0 && s_board[y * BOARD_W + x] == myStone){
        for(int i = 0; i < count; i++){
            s_board[(act_y-1-i) * BOARD_W + act_x+1+i] = myStone;
        }
    }

    // bottom-right
    y = act_y+1;
    x = act_x+1;
    count = 0;
    while(x < BOARD_W && y < BOARD_W && s_board[y * BOARD_W + x] == opponentStone){
        x++;
        y++;
        count ++;
    }
    if(x < BOARD_W && y < BOARD_W && s_board[y * BOARD_W + x] == myStone){
        for(int i = 0; i < count; i++){
            s_board[(act_y+1+i) * BOARD_W + act_x+1+i] = myStone;
        }
    }

    // bottom-left
    y = act_y+1;
    x = act_x-1;
    count = 0;
    while(x >=0 && y < BOARD_W && s_board[y * BOARD_W + x] == opponentStone){
        x--;
        y++;
        count ++;
    }
    if(x >= 0 && y < BOARD_W && s_board[y * BOARD_W + x] == myStone){
        for(int i = 0; i < count; i++){
            s_board[(act_y+1+i) * BOARD_W + act_x-1-i] = myStone;
        }
    }

    // flip the role
    *role = (*role == ROLE::WHITE) ? ROLE::BLACK : ROLE::WHITE;
}


// Every thread calculates one child
// INPUTS:
//  path[i][15:8]: act_x
//  path[i][ 7:0]: act_y
//  children: the action added for each child, same decode as path
// OUTPUTS:
//  win: the number of wins (new results from the simulation) for every node on the path
__global__ simulate_kernel(uint16_t *path, int path_len, 
                           uint16_t *children, int children_len,
                           int *win){

    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int tid = blockDim.x * threadIdx.y + threadIdx.x;

    // shared memory to update the total wins on the path
    __shared__ int s_win[BOARD_W * BOARD_W];
    
    for (int s = 0; tid + s < BOARD_W * BOARD_W; s += blockDim.x * blockDim.y) {
        s_win[tid + s] = 0;
    }
    __syncthreads();

    // every block shares an initial board
    __shared__ uint8_t s_board[BOARD_W * BOARD_W];

    for (int s_y = 0; threadIdx.y + s_y < BOARD_W; s_y += blockDim.y) {
        for (int s_x = 0; threadIdx.x + s_x < BOARD_W; s_x += blockDim.x) {
            int tsy = threadIdx.y + s_y;
            int tsx = threadIdx.x + s_x;
            s_board[tsy * BOARD_W + tsx] = D_NONE;
            if ((threadIdx.y + s_y == 3 && threadIdx.x + s_x == 3) || 
                (threadIdx.y + s_y == 4 && threadIdx.x + s_x == 4)
                s_board[tsy * BOARD_W + tsx] = D_BLACK;
            if ((threadIdx.y + s_y == 3 && threadIdx.x + s_x == 4) || 
                (threadIdx.y + s_y == 4 && threadIdx.x + s_x == 3)
                s_board[tsy * BOARD_W + tsx] = D_WHITE;
        }
    }
    __syncthreads();

    __shared__ ROLE current_role = ROLE::WHITE;

    // Let one thread do all the initialization of the board
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        for (int i = 0; i < path_len; ++i) {
            uint8_t act_x = (uint8_t)(path[i] >> 8) & 0xFFU;
            uint8_t act_y = (uint8_t)path[i] & 0xFFU;

            update_board(s_board, act_x, act_y, &current_role);
        }
    }
    __syncthreads();

    // every thread gets a private copy of the board
    uint8_t p_board[BOARD_W * BOARD_W];
    for (int y = 0; y < BOARD_W; ++y) {
        for (int x = 0; x < BOARD_W; ++x) {
            // TODO: remove bank conflicts
            p_board[y * BOARD_W + x] = s_board[y * BOARD_W + x];
        }
    }

    // update the private board based on the child
    // every thread also gets a private copy of the children
    int actions_len = children_len - 1;
    uint16_t actions[actions_len];
    if (tid < children_len) {
        uint8_t child_x = (uint8_t)(children[tid] >> 8) & 0xFFU;
        uint8_t child_y = (uint8_t)(children[tid]) & 0xFFU;
        update_board(p_board, child_x, child_y, &current_role);
        for (int i = 0; i < children_len - 1; ++i) {
            if (i >= tid) {
                actions[i] = children[i + 1];
            } else {
                actions[i] = children[i];
            }
        }
    }
    __syncthreads();

    // every thread gets a new private ROLE variable
    ROLE p_role = current_role;

    int step = 0;
    while(step < MAX_SIM_STEP){
        step++;
        uint16_t rand_act = git_random_action(actions, &actions_len);
        if (rand_act != 0xFFFFU) {
            uint8_t rand_x = (uint8_t)(rand_act >> 8) & 0xFFU;
            uint8_t rand_y = (uint8_t)(rand_act) & 0xFFU;
            update_board(p_board, rand_x, rand_y, &p_role);
        } else {    // game finishes
            // TODO: get result
        }
    }
    // TODO: draw
}


Action MCTS::run(){
    Board b;    // NOTE: duplicate of the board in main. Can we remove it?

    for(auto action:init_path){
        // initialize the board with history actions
        b.update(action);
    }
    int step = 0;
    while(step < MAX_EXPAND_STEP){
        // cout << "traverse step:" << cstep << endl;
        traverse(root, init_path, b);
        step += 1;
    }
    Action bestMove(0,0);
    double maxv = 0;
    for(auto child : root->children){
        double v = child->score / (child->n + EPSILON);
        if(v >= maxv){
            maxv = v;
            bestMove = child->path.back();
        }
    }
    // get the best move and return
    return bestMove;
}



void MCTS::traverse(Node *root, vector<Action> &path, Board &b){

    stack<Node*> S;
    S.push(root);
    int iter_step = 0;
    while(!S.empty()){
        // cout << iter_step << endl;
        iter_step++;
        Node* node = S.top();
        
        S.pop();
        Node *child = nullptr;
        if(!node->expandable){
            if(node->children.empty()){
                // this is an terminal state
                backprop(node, simulate(node));
            } else{
                S.push(select(node));
            }
        } else{
            node->expandable = false;
            expand(node);

            for(auto child : node->children){   // NOTE: can be parallelized
                backprop(node, simulate(child));
            }
        }
    }
}

Node* MCTS::select(Node* node){
    // cout << "enter select" << endl;
    double maxn = -1;
    Node* child = nullptr;   
    for(auto c : node->children){
        // cout << c << endl;
        double UCB = c->UCB;
        if(UCB > maxn){
            child = c;
            maxn = UCB;
        }
    }
    // cout << child << endl;
    // cout << "exit select" << endl;
    return child;
}

void MCTS::expand(Node * node){
    Board b;
    b.batch_update(node->path);
    vector<Action> actions = b.get_actions();
    // cout << "action size" << actions.size() << endl;
    for(auto action : actions){
        // cout << act_y << act_x << endl;
        node->add_child(new Node(node->path, action));
    }
}

void MCTS::backprop(Node *node, Result result){
        // cout << "enter backprop" << endl;
    bool shouldUpdate = false;
    while(node->parent){
        node = node->parent;
        if(result == Result::WIN) node->score += 1;
        node->n += 1;
        shouldUpdate = !shouldUpdate;   // NOTE: not used?
    }
}


Result MCTS::simulate(Node *root){
    Board b;
    // cout << "enter simulate" << endl;

    for(auto action:root->path){
        b.update(action);
    }
    int step = 0;
    while(step < MAX_SIM_STEP){
        step++;
        if(!rollout(b)){
            return b.get_result();
        }
    }
    return Result::DRAW;
}
bool MCTS::rollout(Board &b){
    vector<Action> actions = b.get_actions();
    if(actions.empty()) return false;
    shuffle(actions.begin(), actions.end(), std::default_random_engine(42));
    b.update(actions[0]);
    return true;
}